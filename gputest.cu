
#include <hip/hip_runtime.h>
#include <emmintrin.h>
#include <sys/time.h>
#include <stdio.h>

const long N = 1000000; // Change array size (may need a long) 

///////////////////////////////////////////////////////////////////////////////////////////////////////////
// HELPER CODE TO INITIALIZE, PRINT AND TIME
struct timeval start, end;


void starttime() {
  gettimeofday( &start, 0 );
}

void endtime(const char* c) {
   gettimeofday( &end, 0 );
   double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
   printf("%s: %f ms\n", c, elapsed); 
}

void init(const char* c) {
  printf("***************** %s **********************\n", c);
  // TMC Commenting Out for Class  
  printf("Running %s...\n", c);
  starttime();
}

void finish(int a, long N, const char* c) {
	endtime(c);
	printf("Done.\n");
	printf("\nThere are %ld Prime numbers between 1 and %ld.", a, N);
	printf("***************************************************\n");
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////

/*
 __global__ void prime(long* a, long high)  {
	// Prime algorithm
	bool check = false;
	for(int i = 2; i <= high/2; ++i) {
		if(high % i == 0) {
			check = true;
			break;
		}
	}
	if(check)
		++a;
}
*/

// Normal C function to square root values
int normal(int a, long N)
{
    long low = 2, high = N, i, check;
	// printf("Prime numbers between 1 and %d are: ",high);
	while (low < high)
	{
		check = 0;
		for(i = 2; i <= low/2; ++i)
		{
		if(low % i == 0)
		{
			check = 1;
			break;
		}
		}
		if (check == 0)
			++a;
		//printf("%d ", low);
		++low;
   }
   return a;
}                                                                                                                                                                                                       

// GPU function to square root values
// Every thread on every core runs this function
__global__ void gpu_prime(int* a, long N) {
   // One element per thread on each core
   // blockIdx.x = Core #
   // blockDim.x = Threads per core
   // threadIdx.x = Thread #
   // The formula below makes sure the value of element 
   // is different on every thread on every core
   long element = blockIdx.x*blockDim.x + threadIdx.x;
   // If there is not an event split, some threads will be 
   // out of bounds
   // We just let those do nothing
   // The rest square root their elements 
	if (element <= N && element >= 2) {
		/*
		if (element % 2 != 0)
			element = N - element;
		//printf("%d\n", element);
		*/
		//printf("%d\n", element);
		int check = 0;	
		for(int i = 2; i <= element/2; ++i) {
        		if(element  % i == 0) {
        		check = 1;
        		break;
        		}
        	}
		if (check == 0){
	                atomicAdd(a,1);	
		}
	}
}

void gpu(int* a, long N) {
   int threadsPerCore = 512; // This can vary, up to 1024
   long numCores = N / threadsPerCore + 1; // This division will work.  If the split is uneven, we overshoot

   // Budget memory for counter
   // Memory must be on the graphics card (use cudaMalloc for this)
   int* gpuA;
   hipMalloc(&gpuA, sizeof(int)); // Allocate enough memory on the GPU
   
   // Copy array of floats a from CPU memory to gpuA on the graphics card
   // Note: This operation is SLOW.  You will have to offset this cost with the parallelism below
   hipMemcpy(gpuA, a, sizeof(int), hipMemcpyHostToDevice); 
   //printf("%ld\n", *gpuA);	
   // Call parallel function with specified number of cores and threads per core
   gpu_prime<<<numCores, threadsPerCore>>>(gpuA, N);

   // Copy square rooted array of floats gpuA from graphics card to a in CPU memory
   // Again, this operation is SLOW.  
   hipMemcpy(a, gpuA, sizeof(int), hipMemcpyDeviceToHost); 
   
   // Release the memory for gpuA
   hipFree(&gpuA); // Free the memory on the GPU
}
                                                                                                                                                                                               
 

int main()                                                                                                                                                                                  
{
	/////////////////////////////////////////////////////////////////////////
	// GPUs will likely have large N
	// Budget memory on the heap, prevent a stack overflow  
	int a = 1;
	/////////////////////////////////////////////////////////////////////////
	
	// Test 1: Sequential For Loop
	init ("Normal");
	a = normal(a, N); 
	finish(a, N, "Normal"); 
	// Test 2: GPU
	a = 1;
	init("GPU");
	gpu(&a, N);  
	finish(a, N, "GPU");

	// Memory on the heap must be freed manually
	//free(&a);
	return 0;
}

